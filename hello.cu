
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5

__global__ void cuda_hello()
{
	printf("Hello world from GPU!\n");
}

__global__ void vector_add(float *out, float *a, float *b, int n)
{
	for (int i = 0; i < n; ++i)
		out[i] = a[i] + b[i];
}

int main()
{
	cuda_hello<<<1, 1>>>();
	printf("Hello world from host\n");

	// float a[N], b[N], out[N];
	float *a, *b, *out;
	hipMalloc((void **)&a, sizeof(float) * N);
	hipMalloc((void **)&b, sizeof(float) * N);
	hipMalloc((void **)&out, sizeof(float) * N);

	for (int i = 0; i < N; ++i)
		a[i] = b[i] = i;

	vector_add<<<1, 1>>>(out, a, b, N);

	for (int i = 0; i < N; ++i)
		printf("%f + %f = %f\n", a[i], b[i], out[i]);

	return 0;
}
