
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5

__global__ void cuda_hello()
{
	printf("Hello world from GPU!\n");
}

__global__ void vector_add(float *out, float *a, float *b, int n)
{
	for (int i = 0; i < n; ++i)
		out[i] = a[i] + b[i];
}

int main()
{
	cuda_hello<<<1, 1>>>();
	printf("Hello world from host\n");

	float a[N], b[N], out[N];
	float *dev_a, *dev_b, *dev_out;

	hipMalloc((void **)&dev_a, N * sizeof(float));
	hipMalloc((void **)&dev_b, N * sizeof(float));
	hipMalloc((void **)&dev_out, N * sizeof(float));

	for (int i = 0; i < N; ++i)
		a[i] = b[i] = i;

	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

	vector_add<<<1, 1>>>(dev_out, dev_a, dev_b, N);

	hipMemcpy(out, dev_out, N * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i)
		printf("%f + %f = %f\n", a[i], b[i], out[i]);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_out);

	return 0;
}
