/*
 * This program measures execution times for 
 * cuFFT convolution for array sizes of 2^N
 * between MIN_SIZE and MAX_SIZE and saves the 
 * data to FILENAME in a format that can be
 * easily read using NumPy. During execution,
 * progress is printed to standard output, along
 * with a measurement of the maximum error to 
 * ensure the computation is being performed 
 * correctly.
 * 
 * compile using nvcc and link cuFFT with
 * -lcufft 
 */

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#define FILENAME "gpu_convolution_performance_results.txt"
#define TRIALS 10
#define MIN_SIZE 1024 // 2^10
#define MAX_SIZE 268435456 // 2^28
#define BATCH 1 // batches in CUFFT plan
#define BLOCK_SIZE 128 // number of threads in a block

double execution_time(int array_size, int trial);

void convolve(hipfftComplex *input, hipfftComplex *filter, int size,
		hipfftHandle *plan);

__global__ void complex_mult_inplace(hipfftComplex *z1, hipfftComplex *z2, int size);

float error(hipfftComplex z, hipfftComplex z0);

int main(void)
{
	FILE *fp;
	fp = fopen(FILENAME, "w");

	fprintf(fp, "## First column is array size, rest are times in milliseconds\n");

	for (int size = MIN_SIZE; size <= MAX_SIZE; size = size * 2)
	{
		fprintf(fp, "%9d", size); // begin line with array size
		for (int trial = 0; trial < TRIALS; ++trial) // perform trials
			fprintf(fp, " %f", execution_time(size, trial));
		fprintf(fp, "\n"); // end line
	}

	fclose(fp);

	return 0;
}

double execution_time(int array_size, int trial)
{
	/*
	   this function measures execution time and checks correctness
	   of a function for performing FFT convolution using CUFFT.
	   the filter used for convolution has an initial element of 
	   1.0 + 0.0i followed by all zeros. this choice was made so that
	   correctness could be checked by comparing the output array to 
	   a copy of the input array.
	*/

	// announce start of trial set
	if (trial == 0)
		printf("%d array elements", array_size);

	// allocate host arrays
	hipfftComplex *input, *filter, *comparison;
	input = (hipfftComplex *)malloc(array_size * sizeof(hipfftComplex));
	filter = (hipfftComplex *)malloc(array_size * sizeof(hipfftComplex));
	comparison = (hipfftComplex *)malloc(array_size * sizeof(hipfftComplex));

	// initialize host arrays
	for (int i = 0; i < array_size; ++i)
	{
		// input and comparison initialized with same values
		input[i].x = comparison[i].x = (float)i;
		input[i].y = comparison[i].y = 0.0f;

		// filter initialized with all zeros
		filter[i].x = filter[i].y = 0.0f;
	}
	// initialize first element as 1.0 + 0.0i
	// (divide by array_size for initialization)
	filter[0].x = 1.0f / (float)array_size;

	// allocate device arrays
	hipfftComplex *dev_input, *dev_filter;
	hipMalloc((void **)&dev_input, array_size * sizeof(hipfftComplex));
	hipMalloc((void **)&dev_filter, array_size * sizeof(hipfftComplex));

	// create cufft plan
	hipfftHandle plan;
	hipfftPlan1d(&plan, array_size, HIPFFT_C2C, BATCH);

	// copy data from host arrays to device arrays
	hipMemcpy(dev_input, input, array_size * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipMemcpy(dev_filter, filter, array_size * sizeof(hipfftComplex), hipMemcpyHostToDevice);

	// measure execution time of convolution
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	convolve(dev_input, dev_filter, array_size, &plan);
	
	hipEventRecord(end);
	hipEventSynchronize(end);
	float milliseconds = 0.0f;
	hipEventElapsedTime(&milliseconds, start, end);

	// copy data from device input array to host input array
	hipMemcpy(input, dev_input, array_size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	// check max error and print results
	float max_error = 0.0f, current_error = 0.0f;
	for (int i = 0; i < array_size; ++i)
	{
		current_error = error(input[i], comparison[i]);
		if (current_error > max_error)
			max_error = current_error;
	}
	printf(" max error: %f;", max_error);
	if (trial == TRIALS - 1)
		printf("\n");

	// clean up
	hipfftDestroy(plan);
	hipFree(dev_input);
	hipFree(dev_filter);
	free(input);
	free(filter);
	free(comparison);

	return milliseconds;
}

void convolve(hipfftComplex *input, hipfftComplex *filter, int size,
		hipfftHandle *plan)
{
	// forward Fourier transform input and filter
	hipfftExecC2C(*plan, input, input, HIPFFT_FORWARD);
	hipfftExecC2C(*plan, filter, filter, HIPFFT_FORWARD);
	//cudaDeviceSynchronize();

	// multiply transformed arrays element-wise
	int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	complex_mult_inplace<<<grid_size, BLOCK_SIZE>>>(input, filter, size);

	// inverse Fourier transform on product array
	hipfftExecC2C(*plan, input, input, HIPFFT_BACKWARD);
}

__global__ void complex_mult_inplace(hipfftComplex *z1, hipfftComplex *z2, int size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < size)
	{
		float tx, ty, x1, y1, x2, y2;

		x1 = z1[tid].x;
		y1 = z1[tid].y;

		x2 = z2[tid].x;
		y2 = z2[tid].y;

		tx = x1 * x2 - y1 * y2;
		ty = x1 * y2 + y1 * x2;

		z1[tid].x = tx;
		z1[tid].y = ty;
	}
}

float error(hipfftComplex z, hipfftComplex z0)
{
	float dx, dy;
	dx = z.x - z0.x;
	dy = z.y - z0.y;

	return sqrtf(dx * dx + dy * dy);
}
