
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 256
#define BATCH 10

int main(void)
{
	hipfftHandle plan;
	hipfftComplex *data;

	hipMalloc((void **)&data, NX * BATCH * sizeof(hipfftComplex));
	hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH);

	hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
	hipDeviceSynchronize();

	hipfftDestroy(plan);
	hipFree(data);

	return 0;
}
